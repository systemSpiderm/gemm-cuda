#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define BLOCK_SIZE 32

//从lower到upper的随机浮点数生成器
double get_random_double(double lower, double upper);

// 随机生成矩阵，100到100000
void initialize_matrix(double* mat, int rows, int cols);

// GPU 核函数，使用共享内存
__global__ void matrix_multiply_shared(double* A, double* B, double* C, int M, int N, int K);
// 矩阵乘法主函数
void perform_matrix_multiply(double* A, double* B, double* C, int M, int N, int K);


int main(int argc, char* argv[]) {
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    hipSetDevice(dev);

    int M, N, K;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    

    if (argc != 4) {
        fprintf(stderr, "The program %s didnot get enough parameters, please enter M, N, K\n", argv[0]);
        exit(1);
    }

    M = atoi(argv[1]);
    N = atoi(argv[2]);
    K = atoi(argv[3]);

    size_t size_A = M * N * sizeof(double);
    size_t size_B = N * K * sizeof(double);
    size_t size_C = M * K * sizeof(double);


    double* A = (double*)malloc(size_A);
    double* B = (double*)malloc(size_B);
    double* C = (double*)malloc(size_C);


    // 随机初始化矩阵
    initialize_matrix(A, M, N);
    initialize_matrix(B, N, K);
    memset(C, 0, size_C);

    hipEventRecord(start, 0);

    perform_matrix_multiply(A, B, C, M, N, K);

    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Matrix multiplication with shared memory time: %.2f ms\n", elapsedTime);
    
    free(A);
    free(B);
    free(C);

    return 0;
}

double get_random_double(double lower, double upper) {
    int random_int = rand();
    return lower + (double)(random_int / (RAND_MAX + 1.0)) * (upper - lower);
}

void initialize_matrix(double *matrix, int M, int N) {
    for (int i = 0; i < M * N; i++) {
        matrix[i] = get_random_double(100.0, 100000.0);
    }
}

void perform_matrix_multiply(double* A, double* B, double* C, int M, int N, int K) {
    double *d_A, *d_B, *d_C;
    size_t size_A = M * N * sizeof(double);
    size_t size_B = N * K * sizeof(double);
    size_t size_C = M * K * sizeof(double);

    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_B, size_B);
    hipMalloc((void**)&d_C, size_C);

    hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((K + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);

    matrix_multiply_shared<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, M, N, K);

    hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}


__global__ void matrix_multiply_shared(double* A, double* B, double* C, int M, int N, int K) {
    __shared__ double shared_A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double shared_B[BLOCK_SIZE][BLOCK_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = blockIdx.y * BLOCK_SIZE + ty;
    int col = blockIdx.x * BLOCK_SIZE + tx;

    double sum = 0.0f;

    // 循环分块计算
    for (int t = 0; t < (N + BLOCK_SIZE - 1) / BLOCK_SIZE; ++t) {
        
        if (row < M && t * BLOCK_SIZE + tx < N) 
            shared_A[ty][tx] = A[row * N + t * BLOCK_SIZE + tx];
        else 
            shared_A[ty][tx] = 0.0f;

        if (col < K && t * BLOCK_SIZE + ty < N) 
            shared_B[ty][tx] = B[(t * BLOCK_SIZE + ty) * K + col];
        else 
            shared_B[ty][tx] = 0.0f;
        
        __syncthreads(); 

        // 计算子块
        for (int i = 0; i < BLOCK_SIZE; ++i) {
            sum += shared_A[ty][i] * shared_B[i][tx];
        }

        __syncthreads(); // 确保共享内存不被提前覆盖
    }

    if (row < M && col < K) {
        C[row * K + col] = sum;
    }
}