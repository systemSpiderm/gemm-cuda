#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

double get_random_double(double lower, double upper);

void initialize_matrix(double *matrix, int M, int N);

int main(int argc, char* argv[]) {
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    hipSetDevice(dev);

    int M, N, K;

    if (argc != 4) {
        fprintf(stderr, "The program %s didnot get enough parameters, please enter M, N, K\n", argv[0]);
        exit(1);
    }

    M = atoi(argv[1]);
    N = atoi(argv[2]);
    K = atoi(argv[3]);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    size_t size_A = M * N * sizeof(double);
    size_t size_B = N * K * sizeof(double);
    size_t size_C = M * K * sizeof(double);


    double* A = (double*)malloc(size_A);
    double* B = (double*)malloc(size_B);
    double* C = (double*)malloc(size_C);
    double *d_A, *d_B, *d_C;

    hipEventRecord(start, 0);

    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_B, size_B);
    hipMalloc((void**)&d_C, size_C);

    hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);

    // 创建 cuBLAS 句柄
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // 设置矩阵乘法参数
    const double alpha = 1.0f; // 标量 alpha
    const double beta = 0.0f;  // 标量 beta

    // C = alpha * A * B + beta * C
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, K, N, &alpha, d_A,
                M, d_B, N, &beta, d_C, M);

    hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);

    // 清理资源
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime = 0;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("hipblasDgemm ends, Elapsed time: %f ms\n", elapsedTime);

    free(A);
    free(B);
    free(C);

    return 0;
}


double get_random_double(double lower, double upper) {
    int random_int = rand();
    return lower + (double)(random_int / (RAND_MAX + 1.0)) * (upper - lower);
}

void initialize_matrix(double *matrix, int M, int N) {
    for (int i = 0; i < M * N; i++) {
        matrix[i] = get_random_double(100.0, 100000);
    }
}